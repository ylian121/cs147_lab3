
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if ((Row < m) && (Col < n)) {
        float Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < k; ++i) {
            Pvalue += A[Row*k+i]*B[i*n+Col];
        }
        C[Row*n+Col] = Pvalue;
    }
    
    
        
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1, 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    mysgemm<<<DimGrid, DimBlock>>>(m,n,k,A,B,C);
	
    /*************************************************************************/
}


